#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }

template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}

// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d = 0;


    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        
        t += deltas[1]; // real delta
        d += weight * t;
        
        ws += weight;
        
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;

}


void composite_rays_train_forward_rgb(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    image += index * 3;
    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;
    grad_sigmas += offset;
    grad_rgbs += offset * 3;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0];

    scalar_t r = 0, g = 0, b = 0, ws = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;

        T *= 1.0f - alpha;
        
        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;


        scalar_t sum = 0; 
        sum += grad_image[0] * (T * rgbs[0] - (r_final - r));
        sum += grad_image[1] * (T * rgbs[1] - (g_final - g));
        sum += grad_image[2] * (T * rgbs[2] - (b_final - b));

        sum += grad_weights_sum[0] * (1 - ws_final);

        grad_sigmas[0] = deltas[0] * sum;

        // write grad_sigmas
        //grad_sigmas[0] = deltas[0] * (
        //    grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
        //    grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
        //    grad_image[2] * (T * rgbs[2] - (b_final - b)) +
        //    grad_weights_sum[0] * (1 - ws_final)
        //);

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
        
        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;

        step++;
    }
}


void composite_rays_train_backward_rgb(const at::Tensor grad_weights_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];




    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t += deltas[1]; // real delta
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
}


void composite_rays_rgb(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}
