#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }

template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}

// sigmas: [M]
// semantics: [M, CLASS_NUM]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ semantics,  
    const uint32_t semantic_class_length,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * semantic_image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        depth[index] = 0;

        for (uint8_t i = 0; i < semantic_class_length; i++) 
        {
          semantic_image[index * 3 + i] = 0;
        }
        return;
    }

    sigmas += offset;
    semantics += offset * semantic_class_length;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t ws = 0, t = 0, d = 0;

    // lets just use 1000 and hope nobody uses more than 1000 classes
    scalar_t semantic_ids[1000] = {0};

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;


        //TODO: make semantic class length a parameter
        for (uint8_t i = 0; i < 3; i++) 
        {
          semantic_ids[i] += weight * semantics[i];
        }

        t += deltas[1]; // real delta
        d += weight * t;
        
        ws += weight;
        
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        semantics += semantic_class_length;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;

    for (uint8_t i = 0; i < 3; i++) 
    {
      semantic_image[index * 3 + i] = semantic_ids[i];
    }
}


void composite_rays_train_forward_semantic(const at::Tensor sigmas, const at::Tensor semantics, const uint32_t semantic_class_length, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor depth, at::Tensor semantic_image ) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), semantics.data_ptr<scalar_t>(), semantic_class_length, deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), semantic_image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// semantics: [M, NUM_CLASS]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// semantic_image: [N, 100]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_semantic_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ semantics, 
    const uint32_t semantic_class_length,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ semantic_image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_semantics
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_semantic_image += index * semantic_class_length;
    weights_sum += index;
    semantic_image += index * semantic_class_length;
    sigmas += offset;
    semantics += offset * semantic_class_length;
    deltas += offset * 2;
    grad_sigmas += offset;
    grad_semantics += offset * semantic_class_length;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t ws_final = weights_sum[0];

    scalar_t semantic_ids_final[1000] ={0};

    for (uint8_t i = 0; i < 3; i++) 
    {
      semantic_ids_final[i] = semantic_image[i];
    }

    scalar_t ws = 0;

    scalar_t semantic_ids[1000] ={0};

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        ws += weight;

        for (uint8_t i = 0; i < semantic_class_length; i++) 
        {
          semantic_ids[i] += weight * semantics[i];
        }

        T *= 1.0f - alpha;
        
        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.

        for (uint8_t i = 0; i < semantic_class_length; i++) 
        {
          grad_semantics[i] = grad_semantic_image[i] * weight;
        }


        scalar_t sum = 0; 

        for (uint8_t i = 0; i < semantic_class_length; i++) 
        {
          sum += grad_semantic_image[i] * (T * semantics[i] - (semantic_ids_final[i] - semantic_ids[i]));
        }

        sum += grad_weights_sum[0] * (1 - ws_final);

        grad_sigmas[0] = deltas[0] * sum;

        // write grad_sigmas
        //grad_sigmas[0] = deltas[0] * (
        //    grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
        //    grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
        //    grad_image[2] * (T * rgbs[2] - (b_final - b)) +
        //    grad_weights_sum[0] * (1 - ws_final)
        //);

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
        
        // locate
        sigmas++;
        semantics += semantic_class_length;
        deltas += 2;
        grad_sigmas++;
        grad_semantics += semantic_class_length;

        step++;
    }
}


void composite_rays_train_backward_semantic(const at::Tensor grad_weights_sum,const at::Tensor grad_semantic_image, const at::Tensor sigmas,const at::Tensor semantics, const uint32_t semantic_class_length,  const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum,const at::Tensor semantic_image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_semantics) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_semantic_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(),grad_semantic_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(),semantics.data_ptr<scalar_t>(), semantic_class_length, deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(),semantic_image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(),grad_semantics.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////



template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ semantics, 
    const uint32_t semantic_class_length,
    const scalar_t* __restrict__ deltas, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* semantic_image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    semantics += n * n_step * semantic_class_length;
    deltas += n * n_step * 2;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    semantic_image += index * semantic_class_length;

    scalar_t t = rays_t[0]; // current ray's t
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];

    scalar_t semantic_ids[1000] = {0};

    for (uint8_t i = 0; i < semantic_class_length; i++) 
    {
      semantic_ids[i] = semantic_image[i];
    }



    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t += deltas[1]; // real delta
        d += weight * t;

        for (uint8_t i = 0; i < 3; i++) 
        {
          semantic_ids[i] += weight * semantics[i];
        }

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        semantics += semantic_class_length;
        deltas += 2;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;

    for (uint8_t i = 0; i < semantic_class_length; i++) 
    {
      semantic_image[i] = semantic_ids[i];
    }
}


void composite_rays_semantic(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor semantics, const uint32_t semantic_class_length, at::Tensor deltas, at::Tensor weights, at::Tensor depth,at::Tensor semantic_image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    semantic_image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(),semantics.data_ptr<scalar_t>(), semantic_class_length, deltas.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(),semantic_image.data_ptr<scalar_t>());
    }));
}
